﻿#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"

void handleError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%d: %s\n", file, line, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(err) (handleError(err, __FILE__, __LINE__))

int main(void) {
    hipDeviceProp_t prop;
    int count = 0;

    // Obtener el número de dispositivos CUDA
    HANDLE_ERROR(hipGetDeviceCount(&count));

    // Iterar sobre cada dispositivo
    for (int i = 0; i < count; i++) {
        HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
        printf(" --- General Information for device %d ---\n", i);
        printf("Name: %s\n", prop.name);
        printf("Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("Clock rate: %d kHz\n", prop.clockRate); // Añadido 'kHz' para mayor claridad
        printf("Device copy overlap: %s\n", prop.deviceOverlap ? "Enabled" : "Disabled");
        printf("Kernel execution timeout: %s\n", prop.kernelExecTimeoutEnabled ? "Enabled" : "Disabled");

        printf(" --- Memory Information for device %d ---\n", i);
        printf("Total global mem: %ld bytes\n", prop.totalGlobalMem); // Añadido 'bytes' para mayor claridad
        printf("Total constant mem: %ld bytes\n", prop.totalConstMem);
        printf("Max mem pitch: %ld bytes\n", prop.memPitch);
        printf("Texture Alignment: %ld bytes\n", prop.textureAlignment);

        printf(" --- MP Information for device %d ---\n", i);
        printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("Shared mem per mp: %ld bytes\n", prop.sharedMemPerBlock);
        printf("Registers per mp: %d\n", prop.regsPerBlock);
        printf("Threads in warp: %d\n", prop.warpSize);
        printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("\n");
    }

    return 0;
}